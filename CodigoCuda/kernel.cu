#include "hip/hip_runtime.h"
#include "kernel.h"

#define TPB 1024
#define ATOMIC 1 // 0 para no usar el atomicAdd

__global__ void sumOfArrayKernel(double *d_sum_total, double *d_A, long int n) {
	const long int idx = threadIdx.x + blockDim.x * blockIdx.x;
	const int s_idx = threadIdx.x;
	__shared__ double s_data[TPB];

	s_data[s_idx] = (idx<n) ? d_A[idx] : 0.0;
	__syncthreads();

	if (s_idx==0) {
		double blockSum = 0.0;
		for (int j = 0; j < blockDim.x; j++) {
			blockSum += s_data[j];
		}
		//printf("Block_%d, blockSum = %lf\n", blockIdx.x, blockSum);
		if (ATOMIC) {
			atomicAdd(d_sum_total, blockSum);
		}
		else {
			*d_sum_total += blockSum; //Resultados no esperados
		}
	}
}

double sumOfArrayGPU(double *A, long int n){
	double *d_A;
	double *d_sum_total;
	double sum_total;
	
	//1. Crear memoria en la GPU
	hipMalloc(&d_sum_total, sizeof(double));
	hipMalloc(&d_A, n * sizeof(double));

	//Inicializamos en cero
	hipMemset(d_sum_total, 0, sizeof(double));

	//2. Copiar memoria (CPU-->GPU)
	hipMemcpy(d_A, A, n * sizeof(double), hipMemcpyHostToDevice);

	//3. Ejecutar funci�n Kernel
	sumOfArrayKernel <<<(n+TPB-1)/TPB,TPB >>> (d_sum_total,d_A,n);
	//sumOfArrayKernel_V2 << <(n + TPB - 1) / TPB, TPB >> > (d_sum_total, d_A, n);

	//4. Copiar memoria (GPU-->CPU)
	hipMemcpy(&sum_total, d_sum_total, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_sum_total);
	hipFree(d_A);
	hipDeviceReset();
	return(sum_total);
}