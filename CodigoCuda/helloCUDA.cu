#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloCUDA(float e){
    printf("Soy el hilo %d del bloque %d con valor e=%f\n",threadIdx.x,blockIdx.x,e);
}

int main(void){
    printf("\nHello World\n");

    helloCUDA<<<3,4>>>(2.5f);

    return(0);
}
