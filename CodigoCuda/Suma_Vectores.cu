#include "hip/hip_runtime.h"
/*
Programa:      Suma_Vectores.cu
Descripci�n:   Dados dos vectores de tama�o N, sumarlos en paralelo y guardar el resultado en un tercer vector.
Actualizaci�n: 09/Jul/2020
*/


#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>

// Funci�n Kernel que se ejecuta en el Device.
__global__ void Suma_vectores(float *c_d,float *a_d,float *b_d, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N){
	  c_d[idx] = a_d[idx] + b_d[idx];
  }	
}

// C�digo principal que se ejecuta en el Host
int main(void){
	float *a_h,*b_h,*c_h; //Punteros a arreglos en el Host 
	float *a_d,*b_d,*c_d;  //Punteros a arreglos en el Device
	const int N = 24;  //N�mero de elementos en los arreglos  (probar 1000000)
	
	size_t size=N * sizeof(float);

	a_h = (float *)malloc(size); // Pedimos memoria en el Host
	b_h = (float *)malloc(size);
	c_h = (float *)malloc(size);//Tambi�n se puede con hipHostMalloc
	
	//Inicializamos los arreglos a,b en el Host
	srand(time(NULL));
	for (int i=0; i<N; i++){
		//a_h[i] = (float)i;b_h[i] = (float)(i+1);
		a_h[i] = rand() % 100 + 1.0;
		b_h[i] = rand() % 100 + 1.0;
	}

	printf("\nArreglo a:\n");
	for (int i=0; i<N; i++) printf("%f ", a_h[i]);
	printf("\n\nArreglo b:\n");
	for (int i=0; i<N; i++) printf("%f ", b_h[i]);
	
	hipMalloc((void **) &a_d,size);   // Pedimos memoria en el Device
	hipMalloc((void **) &b_d,size);
	hipMalloc((void **) &c_d,size);
	
	//Pasamos los arreglos a y b del Host al Device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
	
	//Realizamos el c�lculo en el Device
	int block_size =8;
	int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
	
	Suma_vectores <<< n_blocks, block_size >>> (c_d,a_d,b_d,N);
		
	//Pasamos el resultado del Device al Host
	hipMemcpy(c_h, c_d, size,hipMemcpyDeviceToHost);
	
	//Resultado
	printf("\n\nArreglo c:\n");
	for (int i=0; i<N; i++) printf("%f ", c_h[i]);
	
	printf("\n\nFin del programa...\n");
	//system("pause");
	
	// Liberamos la memoria del Host
	free(a_h); 
	free(b_h); 
	free(c_h); 

	// Liberamos la memoria del Device
	hipFree(a_d); 
	hipFree(b_d); 
	hipFree(c_d); 
	return(0);
}
